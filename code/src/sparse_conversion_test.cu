/*
 * Test file for conversion between sparse and dense matrices
 * Matrices assumed to be generated using generate_sparse_mat.py
 *
 * cuSPARSE assumes matrices are stored in column major order
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include "sparse_conversion.h"
#include "matrix_io.h"

int main(int argc, char * argv[])
{
  struct Matrix mat;
  if (argc != 2){
    printf("usage ./sparse_conversion_test filename\n");
    exit(1);
  }
  char * filename = argv[1];
  int num_elems;
  read_matrix_dims(filename, &mat, &num_elems);
  mat.vals = (float *)calloc(num_elems, sizeof(float));
  read_matrix_vals(filename, &mat, 1);
  print_matrix(&mat);

  // Initialize cusparse library
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  // Call conversion func
  struct SparseMat spm;
  convert_to_sparse(&spm, &mat, handle);
  copyDeviceCSR2Host(&spm, &mat);

  printf("Num rows: %d\n", mat.dims[2]);
  print_sparse_matrix(spm, mat.dims[2]);

  // Free memory
  hipsparseDestroy(handle);
  destroySparseMatrix(&spm);
  destroyMatrix(&mat);
}
