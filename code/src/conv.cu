#include "hip/hip_runtime.h"
/* Functions for carrying out
 * dense and sparse 2D convolutions
 *
 * Matrices are assumed to be stored in row major order
 */

#include <hipDNN.h>
#include "conv.h"
#include "indexing_defs.h"
#include "safe_call_defs.h"
#include "sparse_conversion.h"
#include "matrix_io.h"

void convolve2DDenseProjectImp(struct Matrix * mat,
                struct Kernel * kernel,
                struct Matrix * result)
{
  // Initialize result matrix
  result->dims[0] = mat->dims[0];
  result->dims[1] = mat->dims[1];
  result->dims[2] = mat->dims[2];
  result->dims[3] = mat->dims[3];
  result->is_column_first = mat->is_column_first;

  // Initialize cuda memory and copy in matrix and result to device
  float * d_input;
  float * d_output;
  size_t image_bytes = mat->dims[2] * mat->dims[3] * sizeof(float);
  size_t out_im_bytes = result->dims[2] * result->dims[3] * sizeof(float);
  size_t kernel_bytes = kernel->dims[2] * kernel->dims[3] * sizeof(float);
  CudaSafeCall(hipMalloc(&d_input, image_bytes));
  CudaSafeCall(hipMemcpy(d_input, mat->vals, image_bytes, hipMemcpyHostToDevice));
  CudaSafeCall(hipMalloc(&d_output, out_im_bytes));
  CudaSafeCall(hipMemset(d_output, 0, out_im_bytes));

  if (!kernel->is_on_device)
  {
    printf("Kernel is not on the device. Allocating to constant memory\n");

    CudaSafeCall(hipMalloc(&kernel->vals_device, kernel_bytes));
    CudaSafeCall(hipMemcpy(kernel->vals_device, kernel->vals, kernel_bytes, hipMemcpyHostToDevice));
    kernel->is_on_device = 1;
  }

  // Create kernel dims
  int t_col = MIN(mat->dims[2], 16);
  int t_row = MIN(mat->dims[3], 16);
  int b_col = mat->dims[2] / t_col;
  int b_row = mat->dims[3] / t_row;
  printf("Grid dim: (%d, %d), block dim: (%d, %d)\n", b_row, b_col, t_row, t_col);
  dim3 dimGrid(b_col, b_row);
  dim3 dimBlock(t_col, t_row);

  // Call convolve kernel
  convolve2DKernel<<<dimGrid, dimBlock, kernel_bytes>>>(
                  d_input,
                  kernel->vals_device,
                  d_output,
                  mat->dims[2],
                  mat->dims[3],
                  kernel->dims[2],
                  kernel->dims[3]);
  CudaCheckError();

  // Copy result back to host
  result->vals = (float *)calloc(out_im_bytes, sizeof(float));
  CudaSafeCall(hipMemcpy(result->vals, d_output, out_im_bytes, hipMemcpyDeviceToHost));
  hipFree(d_input);
  hipFree(d_output);
}


// This implementation follows the approach in chapter 7 of
// Programming Massively Parallel Processors
__global__ void convolve2DKernel(float * matrix,
                        float * kernel,
                        float * result,
                        int mat_h,
                        int mat_w,
                        int k_h,
                        int k_w)
{
    int row_o = blockIdx.y * blockDim.y + threadIdx.x;
    int col_o = blockIdx.x * blockDim.x + threadIdx.y;
    int tidx = row_o * (gridDim.y * blockDim.y) + col_o;
    int row_i = row_o - k_h / 2;
    int col_i = col_o - k_w / 2;
    int i, j;

    // Load kernel into shared memory
    extern __shared__ float k_shared[];
    if (tidx < (k_h * k_w))
    {
      k_shared[tidx] = kernel[tidx];
    }
    __syncthreads();

    // Compute convolution
    float out = 0.0f;
    if (row_o < mat_h && col_o < mat_w)
    {
      for (i = 0; i < k_h; i++)
      {
        for (j = 0; j < k_w; j++)
        {
          if (row_i + i >= 0 && col_i + j >= 0 &&
              row_i + i < mat_h && col_i + j < mat_w)
          {
            out += k_shared[index2D(i, j, k_w)] * matrix[index2D(row_i + i, col_i + j, mat_w)];
          }
        }
      }
      result[index2D(row_o, col_o, mat_w)] = out;
    }
}


// This implementation closely follows this excellent tutorial
// http://www.goldsborough.me/cuda/ml/cudnn/c++/2017/10/01/14-37-23-convolutions_with_cudnn/

void convolve2DDense(struct Matrix * mat,
                struct Kernel * kernel,
                struct Matrix * result, // Not initialized
                hipdnnHandle_t cudnn)
{
  //Initialize input, kernel and output descriptors
  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/1,
                                      /*channels=*/1,
                                      /*image_height=*/mat->dims[2],
                                      /*image_width=*/mat->dims[3]));

  int out_height = mat->dims[2];
  int out_width = mat->dims[3];
  int pad_height = kernel->dims[2] / 2;
  int pad_width = kernel->dims[3] / 2;
  result->dims[0] = mat->dims[0];
  result->dims[1] = mat->dims[1];
  result->dims[2] = out_height;
  result->dims[3] = out_width;
  result->is_column_first = mat->is_column_first;

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/1,
                                        /*image_height=*/out_height,
                                        /*image_width=*/out_width));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/pad_height,
                                             /*pad_width=*/pad_width,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION));

  // Initialize kernel descriptor if it is not on the device already
  if (!kernel->is_on_device)
  {
    printf("Kernel is not on the device. Creating kernel descriptor\n");
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel->kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel->kernel_descriptor,
                                         /*dataType=*/HIPDNN_DATA_FLOAT,
                                         /*format=*/HIPDNN_TENSOR_NCHW,
                                         /*out_channels=*/1,
                                         /*in_channels=*/1,
                                         /*kernel_height=*/kernel->dims[2],
                                         /*kernel_width=*/kernel->dims[3]));
  }

  // Describe algorithm
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
    hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                        input_descriptor,
                                        kernel->kernel_descriptor,
                                        convolution_descriptor,
                                        output_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        /*memoryLimitInBytes=*/0,
                                        &convolution_algorithm));

  // Allocate memory on device
  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel->kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));

  void * d_workspace;
  CudaSafeCall(hipMalloc(&d_workspace, workspace_bytes));
  size_t image_bytes = mat->dims[2] * mat->dims[3] * sizeof(float);
  size_t out_im_bytes = result->dims[2] * result->dims[3] * sizeof(float);
  size_t kernel_bytes = kernel->dims[2] * kernel->dims[3] * sizeof(float);
  float * d_input;
  float * d_output;

  CudaSafeCall(hipMalloc(&d_input, image_bytes));
  CudaSafeCall(hipMemcpy(d_input, mat->vals, image_bytes, hipMemcpyHostToDevice));
  CudaSafeCall(hipMalloc(&d_output, out_im_bytes));
  CudaSafeCall(hipMemset(d_output, 0, out_im_bytes));

  if (!kernel->is_on_device)
  {
    printf("Kernel is not on the device. Allocating memory\n");
    CudaSafeCall(hipMalloc(&kernel->vals_device, kernel_bytes));
    CudaSafeCall(hipMemcpy(kernel->vals_device, kernel->vals, kernel_bytes, hipMemcpyHostToDevice));
    kernel->is_on_device = 1;
  }

  // Convolve
  const float alpha = 1, beta = 0;
  checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     d_input,
                                     kernel->kernel_descriptor,
                                     kernel->vals_device,
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     d_workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     d_output));

  // Copy result back to host
  result->vals = (float *)calloc(out_im_bytes, sizeof(float));
  CudaSafeCall(hipMemcpy(result->vals, d_output, out_im_bytes, hipMemcpyDeviceToHost));
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
}


void destroyKernel(struct Kernel * kernel, struct Matrix * kernel_mat)
{
  hipdnnDestroyFilterDescriptor(kernel->kernel_descriptor);
  hipFree(kernel->vals_device);
  destroyMatrix(kernel_mat);
}


void convolve2DSparse(struct SparseMat * mat,
                struct Kernel * kernel,
                struct SparseMat * result)
{
  // TODO
  // Copy kernel to device if not there - constant memory
  // Put matrix, and sparse mat result in constant memory
  // Convolve (look up csr indexing)
  // Copy back
}
